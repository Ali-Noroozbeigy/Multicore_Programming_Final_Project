#include "hip/hip_runtime.h"
#include "stdlib.h"
#include "stdio.h"
#include "time.h"
#include "string.h"
#include "hip/hip_runtime.h"
#include ""

#define NUM_BLOCKS 256
#define NUM_THREADS 256
#define NUM_CLASSES 6

hipError_t compute_histogram(int* histo, char** input, int n);
void print_array(int* a);

__device__ int cuda_strcmp(const char* s1, const char* s2) {
	while (*s1 && *s2 && (*s1 == *s2)) {
		s1++;
		s2++;
	}
	return (*s1 - *s2);
}

__global__ void compute_hist_kernel(int* histo, char** input, int n) {

	__shared__ int histo_private[NUM_CLASSES];

	if (threadIdx.x < NUM_CLASSES) histo_private[threadIdx.x] = 0;
	__syncthreads();

	int i = threadIdx.x + blockIdx.x * blockDim.x;

	int stride = blockDim.x * gridDim.x;

	while (i < n) {
	
		char *s1 = "elephant";
		char *s2 = input[i];
		while (*s1 && *s2 && (*s1 == *s2)) {
			s1++;
			s2++;
		}
		if (*s1 - *s2 == 0) atomicAdd(&(histo_private[0]), 1);
		
		s1 = "lion";
		while (*s1 && *s2 && (*s1 == *s2)) {
			s1++;
			s2++;
		}
		if (*s1 - *s2 == 0) atomicAdd(&(histo_private[1]), 1);
		
		s1 = "zebra";
		while (*s1 && *s2 && (*s1 == *s2)) {
			s1++;
			s2++;
		}
		if (*s1 - *s2 == 0) atomicAdd(&(histo_private[2]), 1);
		
		s1 = "monkey";
		while (*s1 && *s2 && (*s1 == *s2)) {
			s1++;
			s2++;
		}
		if (*s1 - *s2 == 0) atomicAdd(&(histo_private[3]), 1);		

		s1 = "tiger";
		while (*s1 && *s2 && (*s1 == *s2)) {
			s1++;
			s2++;
		}
		if (*s1 - *s2 == 0) atomicAdd(&(histo_private[4]), 1);
		
		
		s1 = "leopard";
		while (*s1 && *s2 && (*s1 == *s2)) {
			s1++;
			s2++;
		}
		if (*s1 - *s2 == 0) atomicAdd(&(histo_private[5]), 1);

		i += stride;
		
	}

	__syncthreads();
	if (threadIdx.x < NUM_CLASSES) atomicAdd(&(histo[threadIdx.x]), histo_private[threadIdx.x]);
}

int main() {
	const char* words[] = { "elephant", "lion", "zebra", "monkey", "tiger", "leopard" };

	int n = 4;

	printf("Please enter number of entries in file: \n");
	//scanf("%d\n", &n);

	// allocate memory for input
	char** input = (char**)malloc(n * sizeof(char*));

	if (input == NULL) {
		printf("Failed to allocate memory");
		return 1;
	}

	// randomly select from words array to fill input n times
	srand(42);
	for (int i = 0; i < n; i++) {
		int index = rand() % NUM_CLASSES;
		input[i] = strdup(words[index]);
	}

	// because number of input is large, in order to check correctness of parallel algorithm,
	// we first calculate hostogram seral
	int count_words_without_parallel[NUM_CLASSES];
	memset(count_words_without_parallel, 0, sizeof(count_words_without_parallel));

	for (int i = 0; i < n; i++) {
		for (int j = 0; j < NUM_CLASSES; j++) {
			if (strcmp(input[i], words[j]) == 0) {
				count_words_without_parallel[j]++;
				break;
			}
		}
	}

	printf("count of each word in serial:\n");
	print_array(count_words_without_parallel);

	int global_histo[NUM_CLASSES];
	for (int i = 0; i < NUM_CLASSES; i++) global_histo[i] = 0;

	compute_histogram(global_histo, input, n);

	print_array(global_histo);

	return EXIT_SUCCESS;

}

hipError_t compute_histogram(int* histo, char** input, int n) {
	char** dev_input = 0;
	int* dev_histo = 0;
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		printf("hipSetDevice failed! Do you have a CUDA-capable GPU installed?");
	}

	cudaStatus = hipMalloc((void**)&dev_histo, n * sizeof(int));
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
	}

	cudaStatus = hipMalloc((void**)&dev_input, n * sizeof(char*));
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
	}

	cudaStatus = hipMemcpy(dev_input, input, n * sizeof(char*), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy failed!");
	}

	clock_t start = clock();

	compute_hist_kernel << <NUM_BLOCKS, NUM_THREADS >> > (dev_histo, dev_input, n);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		printf("addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		printf("hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
	}


	double elapsedtime = (double)(clock() - start) / CLOCKS_PER_SEC;

	printf("elapsed time: %f\n", elapsedtime);

	cudaStatus = hipMemcpy(histo, dev_histo, n * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy failed!");
	}

	hipFree(dev_histo);
	hipFree(dev_input);

	return cudaStatus;
}

void print_array(int* a) {
	int i;
	printf("[-] histogram: ");
	for (i = 0; i < NUM_CLASSES; ++i) {
		printf("%d, ", a[i]);
	}
	printf("\b\b  \n");
}
